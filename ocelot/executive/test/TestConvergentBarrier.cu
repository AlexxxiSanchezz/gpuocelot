/*!
	\file TestConvergentBarrier.cu
	\date 31 January 2012
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates convergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void convergence(int *A) {
	int i = threadIdx.x;
	__shared__ float x[4];
	x[threadIdx.x] = A[i];
	__syncthreads();
	A[i] = x[3 - threadIdx.x] * 2;
}

static void testConvergence() {
	const int N = 4;
	int *A_host, *A_device;
	size_t bytes = N * sizeof(int);
	
	A_host = (int *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (int)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	convergence<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		int expected = (4-(i)) * 2;
		int got = A_host[i];
		if (expected != got) {
			++errors;
			printf("error [%d] - expected: %d, got %d\n", i, expected, got);
		}
	}
	
	free(A_host);
	hipFree(A_device);
	
	printf("Pass/Fail : %s\n", (errors ? "Fail":"Pass"));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	testConvergence();

	return 0;
}
